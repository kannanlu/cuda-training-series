
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> // For rand()
#include <time.h>   // For time()
#define N 1024

// Error checking macro
#define cudaCheckError()                                       \
    {                                                          \
        hipError_t e = hipGetLastError();                    \
        if (e != hipSuccess)                                  \
        {                                                      \
            printf("CUDA error: %s\n", hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    }

// random array population
void random_ints(int *array, int size)
{
    for (int ii = 0; ii < size; ii++)
    {
        array[ii] = rand() % 100;
    }
}

// cuda addition
__global__ void add(int *a, int *b, int *c, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size)
    {
        c[index] = a[index] + b[index];
    }
}

int main(void)
{
    int *a, *b, *c;       // host copies of a b c
    int *d_a, *d_b, *d_c; // device copies of a b c
    int size = N * sizeof(int);
    clock_t t0, t1, t2, t3, t4; // for timing

    t0 = clock();

    // device memory allocation
    hipMalloc((void **)&d_a, size);
    cudaCheckError();
    hipMalloc((void **)&d_b, size);
    cudaCheckError();
    hipMalloc((void **)&d_c, size);
    cudaCheckError();

    // host memory allocation
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // host data creation
    // constant seed
    srand(0);
    // time varying seed
    // srand(time(NULL));
    random_ints(a, N);
    random_ints(b, N);

    t1 = clock();
    printf("Init took %f seconds.\n", ((double)(t1 - t0)) / CLOCKS_PER_SEC);

    // data transfer from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    cudaCheckError();

    t2 = clock();

    // perform addition at device
    int threadsPerBlock = 128; // max is 1024
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    cudaCheckError();
    t3 = clock();

    // transfer result to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    cudaCheckError();

    t4 = clock();
    printf("Data transfer took %f seconds.\n", ((double)(t2 - t1 + t4 - t3)) / CLOCKS_PER_SEC);
    printf("Compute took %f seconds.\n", ((double)(t3 - t2)) / CLOCKS_PER_SEC);

    // print result
    // printf("result is\n");
    // for (int ii = 0; ii < N; ii++)
    // {
    //     printf(" %d", c[ii]);
    // }
    // printf("\n");
    printf("a[0] = %d", a[0]);
    printf(" a[1] = %d\n", a[1]);
    printf("b[0] = %d b[1] = %d\n", b[0], b[1]);
    printf("c[0] = %d c[1] = %d\n", c[0], c[1]);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}